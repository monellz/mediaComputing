
#include <hip/hip_runtime.h>
#define dis2(a_x, a_y, b_x, b_y) (((a_x) - (b_x)) * ((a_x) - (b_x)) + ((a_y) - (b_y)) * ((a_y) - (b_y)))
#define normalize(x) ((x) > 1.0? 1.0: ((x) < 0.0? 0.0: (x)))

extern "C" __device__ double half_tan(
    unsigned a_x, unsigned a_y,
    unsigned b_x, unsigned b_y,
    unsigned c_x, unsigned c_y
) {
    double ac2 = dis2((double)a_x, (double)a_y, (double)c_x, (double)c_y);
    double bc2 = dis2((double)b_x, (double)b_y, (double)c_x, (double)c_y);
    double ab2 = dis2((double)a_x, (double)a_y, (double)b_x, (double)b_y);

    double cos = (ab2 + bc2 - ac2) / (2.0 * sqrt(ab2) * sqrt(bc2));
    if (cos > 1.0) { cos = 1.0; }
    else if (cos < -1.0) { cos = -1.0; }

    return sqrt((1.0 - cos) / (1.0 + cos));
}

extern "C" __global__ void mvc_interpolant(
    //input
    const unsigned int* fg_mask_idx_x,
    const unsigned int* fg_mask_idx_y,
    const unsigned int* fg_bound_idx_x,
    const unsigned int* fg_bound_idx_y,
    const double* fg_bound_r,
    const double* fg_bound_g,
    const double* fg_bound_b,

    const unsigned int len,
    const unsigned int bound_len,

    //output
    double* out_r,
    double* out_g,
    double* out_b
) {
    //data parallel
    double sum;
    unsigned prev_x, prev_y, next_x, next_y;
    double reg_r, reg_g, reg_b;
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
        sum = reg_r = reg_g = reg_b = 0.0;
        for (int j = 0; j < bound_len; ++j) {
            if (j == 0) { prev_x = fg_bound_idx_x[bound_len - 1]; prev_y = fg_bound_idx_y[bound_len - 1]; }
            else { prev_x = fg_bound_idx_x[j - 1]; prev_y = fg_bound_idx_y[j - 1]; }

            if (j == bound_len - 1) { next_x = fg_bound_idx_x[0]; next_y = fg_bound_idx_y[0]; }
            else { next_x = fg_bound_idx_x[j + 1]; next_y = fg_bound_idx_y[j + 1]; }

            double w = (half_tan(prev_x, prev_y, fg_mask_idx_x[i], fg_mask_idx_y[i], fg_bound_idx_x[j], fg_bound_idx_y[j])
                        + half_tan(fg_bound_idx_x[j], fg_bound_idx_y[j], fg_mask_idx_x[i], fg_mask_idx_y[i], next_x, next_y))
                        / sqrt(dis2((double)fg_bound_idx_x[j], (double)fg_bound_idx_y[j], (double)fg_mask_idx_x[i], (double)fg_mask_idx_y[i]));

            sum += w;
            reg_r += w * fg_bound_r[j];
            reg_g += w * fg_bound_g[j];
            reg_b += w * fg_bound_b[j];
        }

        out_r[i] = reg_r / sum;
        out_g[i] = reg_g / sum;
        out_b[i] = reg_b / sum;
    }
}